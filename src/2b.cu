
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>	
#include <time.h>


__global__ void euler_step(float * array, int m, int step) {
	float dt = powf(10,-3);
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < m) {
		array[tId] = array[tId] + dt*(4*(dt*step)-array[tId]+3+tId);
	};
};




int main() {
	hipEvent_t start, stop;
	int e_s = 1000;
	int n_i = 1000;
	int block_size = 256;
	for(int m =0; m < 5;m++){
		e_s = e_s*10;
		float elapsed=0;
  		int grid_size = (int) ceil((float)e_s / block_size);
		float * resultados = (float *) malloc(e_s * sizeof(float));
		float * d_r;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		for(int m = 0; m < e_s; m++){
			resultados[m] = m;
		}
		hipMalloc(&d_r, e_s * sizeof(float));
		hipMemcpy(d_r, resultados, e_s * sizeof(float), hipMemcpyHostToDevice);

		hipEventRecord(start, 0);
		for(int n = 0; n < n_i; n++){
			euler_step<<<grid_size, block_size>>>(d_r,e_s,n);
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		hipMemcpy(resultados, d_r, e_s  * sizeof(float), hipMemcpyDeviceToHost);
		
		printf("Executed with %d equations\n", e_s);
		printf("The elapsed time in gpu was %.2f ms \n", elapsed);
		//printf("%f\n", resultados[0]);

		free(resultados);
		hipFree(d_r);
	}




return 0;

}
