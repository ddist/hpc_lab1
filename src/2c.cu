
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>	
#include <time.h>
#include <math.h>


#define E 2.71828182845904523536


__global__ void euler_step(float * array, int m, int step) {
	float dt = powf(10,-3);
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < m) {
		array[tId] = array[tId] + dt*(4*(dt*step)-array[tId]+3+tId);
	};
};




int main() {
	hipEvent_t start, stop;
	int e_s = 100000000;
	int n_i = 1000;
	int block[4] = {64,128,256,512};
	for(int m =0; m < 4;m++){
		float error = 0;
		int block_size = block[m];
		float elapsed=0;
  		int grid_size = (int) ceil((float)e_s / block_size);
		float * resultados = (float *) malloc(e_s * sizeof(float));
		float * d_r;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		for(int k = 0; k < e_s; k++){
			resultados[k] = k;
		}
		hipMalloc(&d_r, e_s * sizeof(float));
		hipMemcpy(d_r, resultados, e_s * sizeof(float), hipMemcpyHostToDevice);

		hipEventRecord(start, 0);
		for(int n = 0; n < n_i; n++){
			euler_step<<<grid_size, block_size>>>(d_r,e_s,n);
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		hipMemcpy(resultados, d_r, e_s  * sizeof(float), hipMemcpyDeviceToHost);

		for(int g = 0; g < e_s; g++){
			error = error + powf(resultados[g]-((1/E)+4-1+g),2);
		}
		
		printf("Executed with %d blocks\n", block[m]);
		printf("The elapsed time in gpu was %.2f ms \n", elapsed);
		printf("Mean squared error: %f \n", error/e_s);		

		free(resultados);
		hipFree(d_r);
	}




return 0;

}

