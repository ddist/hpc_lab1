
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define E 2.71828182845904523536

__global__ void euler_gpu_2(float * array, float * suma, float dt, int n){
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < n) {
		array[tId] = -1 + dt*suma[tId];
	};
};

int main(int argc, char const *argv[]){
	hipEvent_t start, stop;
	clock_t t1, t2;
	float dts[6] = {0.1, 0.01, 0.001, 0.0001, 0.00001, 0.000001};
    int block_size = 256;
    for(size_t i = 0; i < 6; i++){
    	int n = (int)(10/dts[i]);
    	int grid_size = (int) ceil((float)n / block_size);
    	float elapsed=0;
    	double cpu_time = 0;
    	double error = 0;
    	float * resultados = (float *) malloc(n * sizeof(float));
    	float * sumatoria = (float *) malloc(n * sizeof(float));
    	float * d_r;
    	float * d_s;
    	sumatoria[0] = 1;
    	t1 = clock(); 
    	for(int j =1; j < n; j++){
    		sumatoria[j] = powf(E, -dts[i]*j) + sumatoria[j-1];
    	}
    	t2 = clock();
    	hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		hipMalloc(&d_r, n * sizeof(float));
		hipMalloc(&d_s, n * sizeof(float));
		hipMemcpy(d_r, resultados, n * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_s, sumatoria, n * sizeof(float), hipMemcpyHostToDevice);
		euler_gpu_2<<<grid_size, block_size>>>(d_r,d_s, dts[i], n);
		hipMemcpy(resultados, d_r, n  * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(sumatoria, d_s, n  * sizeof(float), hipMemcpyDeviceToHost);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		for(int g = 0; g < n; g++){
			float real =  -powf(E, -dts[i]*g);
			error = error + powf((resultados[g]-real),2);
		}
		cpu_time = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
		printf("Executed with %f dt\n", dts[i]);
		printf("Mean squared error: %.16f \n", error/n);
		printf("The elapsed time in gpu was %.2f ms \n", elapsed);
		printf("The elapsed time in cpu was %.2f ms \n", cpu_time);
		printf("The total time was %.2f ms \n", elapsed + cpu_time);
	}

	return 0;

}

