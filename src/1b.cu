
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define E 2.71828182845904523536


__global__ void euler_gpu(float * array, float y0, float dt, int n){
    // Initial condition
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if (tId < n) {
    	float initial_value = y0; 
	    for(size_t i = 1; i < tId; i++){
	        initial_value = initial_value + dt*powf(E, -dt*i);
	    };
	    array[tId] = initial_value;

	};
};



int main(int argc, char const *argv[]){
	hipEvent_t start, stop;
	float dts[6] = {0.1, 0.01, 0.001, 0.0001, 0.00001, 0.000001};
    float y0 = -1.0;
    int block_size = 256;

    for(size_t i = 0; i < 6; i++){
    	int n = (int)(10/dts[i]);
    	float elapsed=0;
    	double error = 0;

    	float * resultados = (float *) malloc(n * sizeof(float));
    	float * d_r;

    	int grid_size = (int) ceil((float)n / block_size);

    	hipEventCreate(&start);
		hipEventCreate(&stop);

		hipMalloc(&d_r, n * sizeof(float));
		hipMemcpy(d_r, resultados, n * sizeof(float), hipMemcpyHostToDevice);
		hipEventRecord(start, 0);
		euler_gpu<<<grid_size, block_size>>>(d_r, y0, dts[i], n);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		hipMemcpy(resultados, d_r, n  * sizeof(float), hipMemcpyDeviceToHost);
		for(int g = 0; g < n; g++){
			float real =  -powf(E, -dts[i]*g);
			error = error + powf((resultados[g]-real),2);
		}

		printf("Executed with %f dt\n", dts[i]);
		printf("The elapsed time in gpu was %.2f ms \n", elapsed);
		printf("Mean squared error: %.16f \n", error/n);		

		free(resultados);
		hipFree(d_r);
    }
    return 0;
};



